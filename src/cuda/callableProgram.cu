#include "hip/hip_runtime.h"
#include "CUDA.hpp"

/* change spherical vector into normalized xyz vector*/
__forceinline__ __host__ __device__ vec3f sphere_to_normal(const vec3f &sphere_direction)
{
    const float &theta = sphere_direction.y;
    const float &phi = sphere_direction.z;
    return vec3f(cosf(theta) * sinf(phi),
                 sinf(theta) * sinf(phi),
                 cosf(phi));
}

/* change xyz vector into spherical vector with 0 length */
// __forceinline__ __host__ __device__ vec3f normal_to_sphere(const vec3f &xyz_direction)
// {
//     float theta = atan(xyz_direction.y / xyz_direction.x);
//     float phi = acos(xyz_direction.z / length(xyz_direction));
//     if (xyz_direction.x < 0)
//     {
//         theta = (xyz_direction.y > 0) ? theta + M_PI : theta - M_PI;
//     }
//     return vec3f(0.f, theta, phi);
// }

/* change screen space position into ray direction */
__forceinline__ __host__ __device__ vec3f screen_to_direction(const vec2f &screen,
                                                              const vec3f &direction,
                                                              const vec3f &horizontal,
                                                              const vec3f &vertical)
{
    return normalize(direction + (screen.x - 0.5f) * horizontal + (screen.y - 0.5f) * vertical);
}

/* change polor into xy coordinate vector */
__forceinline__ __host__ __device__ vec2f polor_to_normal(const float &theta, const float &r)
{
    return vec2f(cosf(theta) * r, sinf(theta) * r);
}

/* make ray_stencil[] into an array stored directions of a ray stencil.
*ray_stencil: pointer of a vec2f ray directions array (screen space),
center_pixel_location: location of center pixel (screen space),
h: radius (screen space),
N: number of circles,
n: number of rays in first circle, must be multiple of 4. number doubles each time on next circle */
__host__ __device__ void makeRayStencil1(vec2f *const ray_stencil,
                                         const vec2f &center_pixel_location,
                                         const float &h,
                                         const int &N,
                                         const int &n)
{
    assert(h > 0.f);
    assert(N > 0);
    assert(n > 0 && n % 4 == 0);

    float temp_r;
    int temp_n{n};
    float theta;
    int index{0};
    for (int i = 0; i < N; i++)
    {
        temp_r = h / N * (i + 1);
        for (int j = 0; j < temp_n; j++)
        {
            theta = 2 * M_PI * j / temp_n;
            ray_stencil[index + j] = polor_to_normal(temp_r, theta) + center_pixel_location;
        }
        index += temp_n;
        temp_n *= 2;
    }
}

extern "C" __device__ vec3f __continuation_callable__fast_launch(const int &ix, const int &iy)
{
    const int &accumID = optixLaunchParams.frame.accumID;
    const int &numPixelSamples = optixLaunchParams.parameters.NUM_PIXEL_SAMPLES;
    const auto &camera = optixLaunchParams.camera;

    PRD prd;
    prd.random.init(ix + accumID * optixLaunchParams.frame.size.x,
                    iy + accumID * optixLaunchParams.frame.size.y);
    prd.pixelColor = vec3f(255.f);

    // the values we store the PRD pointer in:
    uint32_t u0, u1;
    packPointer(&prd, u0, u1);

    vec3f pixelColor = 0.f;
    for (int sampleID = 0; sampleID < numPixelSamples; sampleID++)
    {
        vec3f rayDir;
        if (camera.camera_type == PINHOLE)
        {
            // normalized screen plane position, in [0,1]^2
            vec2f screen;
            if (numPixelSamples > 1)
            {
                screen = (vec2f(ix + prd.random(), iy + prd.random()) / vec2f(optixLaunchParams.frame.size));
            }
            else
            {
                screen = (vec2f(ix + 0.5f, iy + 0.5f) / vec2f(optixLaunchParams.frame.size));
            }

            // generate ray direction
            rayDir = screen_to_direction(screen, camera.direction, camera.horizontal, camera.vertical);
        }
        else if (camera.camera_type == ENV)
        {
            // sperical coordinate position
            vec3f spherical_position;
            if (numPixelSamples > 1)
            {
                spherical_position = ((ix + prd.random()) * camera.horizontal + (iy + prd.random()) * camera.vertical);
            }
            else
            {
                spherical_position = ((ix + 0.5f) * camera.horizontal + (iy + 0.5f) * camera.vertical);
            }
            spherical_position -= vec3f(0.f, M_PI, 0.f);
            // change into xyz coordinate position
            const vec3f xyz_position(sphere_to_normal(spherical_position));
            // view port transform
            rayDir = {dot(camera.matrix.vx, xyz_position),
                      dot(camera.matrix.vy, xyz_position),
                      dot(camera.matrix.vz, xyz_position)};
        }

        const int &ray_type = optixLaunchParams.parameters.LAUNCH_RAY_TYPE;
        optixTrace(optixLaunchParams.traversable,
                   camera.position,
                   rayDir,
                   0.f,   // tmin
                   1e20f, // tmax
                   0.0f,  // rayTime
                   OptixVisibilityMask(255),
                   OPTIX_RAY_FLAG_DISABLE_ANYHIT, //OPTIX_RAY_FLAG_NONE,
                   ray_type,                      // SBT offset
                   RAY_TYPE_COUNT,                // SBT stride
                   ray_type,                      // missSBTIndex
                   u0, u1);
        pixelColor += prd.pixelColor;
    }
    return pixelColor;
}

extern "C" __device__ vec3f __continuation_callable__classic_launch(const int &ix, const int &iy)
{
    const auto &camera = optixLaunchParams.camera;
    const int &numPixelSamples = optixLaunchParams.parameters.NUM_PIXEL_SAMPLES;
    const vec2i &quality = optixLaunchParams.parameters.RAY_STENCIL_QUALITY;
    const float &radius = optixLaunchParams.parameters.RAY_STENCIL_RADIUS;

    // normalized screen plane position, in [0,1]^2
    vec2f screen;
    screen = (vec2f(ix + 0.5f, iy + 0.5f) / vec2f(optixLaunchParams.frame.size));

    vec3f rayDir = screen_to_direction(screen, camera.direction, camera.horizontal, camera.vertical);

    // // just try to calculate (2^N - 1) * n, cuda seems doesn't suppor pow(int, int)
    // int array_size{1};
    // for (int i = 0; i < quality.x; i++)
    // {
    //     array_size *= 2;
    // }
    // array_size -= 1;
    // array_size *= quality.y;

    // vec2f *rayStencil;
    // rayStencil = new vec2f[array_size];
    // makeRayStencil1(rayStencil, screen, radius, quality.x, quality.y);

    // // generate ray direction
    // vec3f *rayDir;
    // rayDir = new vec3f[array_size + 1];
    // for (int i = 0; i < array_size; i++)
    // {
    //     rayDir[i] = screen_to_direction(rayStencil[i], camera.direction, camera.horizontal, camera.vertical);
    // }
    // rayDir[array_size] = screen_to_direction(screen, camera.direction, camera.horizontal, camera.vertical);

    // // to do: ray tracing..

    // delete[] rayStencil;
    // delete[] rayDir;
    return vec3f(255.f, 255.f, 255.f);
}
