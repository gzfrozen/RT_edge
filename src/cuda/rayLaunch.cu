#include "hip/hip_runtime.h"
#include "CUDA.hpp"

/* change spherical vector into normalized xyz vector*/
__forceinline__ __host__ __device__ vec3f sphere_to_normal(const vec3f &sphere_direction)
{
    const float &theta = sphere_direction.y;
    const float &phi = sphere_direction.z;
    return vec3f(cos(theta) * sin(phi),
                 sin(theta) * sin(phi),
                 cos(phi));
}

/* change xyz vector into spherical vector with 0 length */
// __forceinline__ __host__ __device__ vec3f normal_to_sphere(const vec3f &xyz_direction)
// {
//     float theta = atan(xyz_direction.y / xyz_direction.x);
//     float phi = acos(xyz_direction.z / length(xyz_direction));
//     if (xyz_direction.x < 0)
//     {
//         theta = (xyz_direction.y > 0) ? theta + M_PI : theta - M_PI;
//     }
//     return vec3f(0.f, theta, phi);
// }

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame()
{
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const int accumID = optixLaunchParams.frame.accumID;
    const auto &camera = optixLaunchParams.camera;

    PRD prd;
    prd.random.init(ix + accumID * optixLaunchParams.frame.size.x,
                    iy + accumID * optixLaunchParams.frame.size.y);
    prd.pixelColor = vec3f(255.f);

    // the values we store the PRD pointer in:
    uint32_t u0, u1;
    packPointer(&prd, u0, u1);

    int numPixelSamples = optixLaunchParams.parameters.NUM_PIXEL_SAMPLES;

    vec3f pixelColor = 0.f;
    for (int sampleID = 0; sampleID < numPixelSamples; sampleID++)
    {
        vec3f rayDir;
        if (camera.camera_type == PINHOLE)
        {
            // normalized screen plane position, in [0,1]^2
            vec2f screen;
            if (optixLaunchParams.parameters.NUM_PIXEL_SAMPLES > 1)
            {
                screen = (vec2f(ix + prd.random() - 0.5f, iy + prd.random() - 0.5f) / vec2f(optixLaunchParams.frame.size));
            }
            else
            {
                screen = (vec2f(ix, iy) / vec2f(optixLaunchParams.frame.size));
            }

            // generate ray direction
            rayDir = normalize(camera.direction + (screen.x - 0.5f) * camera.horizontal + (screen.y - 0.5f) * camera.vertical);
        }
        else if (camera.camera_type == ENV)
        {
            // sperical coordinate position
            vec3f spherical_position;
            if (optixLaunchParams.parameters.NUM_PIXEL_SAMPLES > 1)
            {
                spherical_position = ((ix + prd.random() - 0.5f) * camera.horizontal + (iy + prd.random() - 0.5f) * camera.vertical);
            }
            else
            {
                spherical_position = ((float)ix * camera.horizontal + (float)iy * camera.vertical);
            }
            spherical_position -= vec3f(0.f, M_PI, 0.f);
            // change into xyz coordinate position
            const vec3f xyz_position(sphere_to_normal(spherical_position));
            // view port transform
            rayDir = {dot(camera.matrix.vx, xyz_position),
                      dot(camera.matrix.vy, xyz_position),
                      dot(camera.matrix.vz, xyz_position)};
        }

        const int &ray_type = optixLaunchParams.parameters.LAUNCH_RAY_TYPE;
        optixTrace(optixLaunchParams.traversable,
                   camera.position,
                   rayDir,
                   0.f,   // tmin
                   1e20f, // tmax
                   0.0f,  // rayTime
                   OptixVisibilityMask(255),
                   OPTIX_RAY_FLAG_DISABLE_ANYHIT, //OPTIX_RAY_FLAG_NONE,
                   ray_type,                      // SBT offset
                   RAY_TYPE_COUNT,                // SBT stride
                   ray_type,                      // missSBTIndex
                   u0, u1);
        pixelColor += prd.pixelColor;
    }

    const int r = int(255.99f * min(pixelColor.x / numPixelSamples, 1.f));
    const int g = int(255.99f * min(pixelColor.y / numPixelSamples, 1.f));
    const int b = int(255.99f * min(pixelColor.z / numPixelSamples, 1.f));

    // convert to 32-bit rgba value (we explicitly set alpha to 0xff
    // to make stb_image_write happy ...
    const uint32_t rgba = 0xff000000 | (r << 0) | (g << 8) | (b << 16);

    // and write to frame buffer ...
    const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x;
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}