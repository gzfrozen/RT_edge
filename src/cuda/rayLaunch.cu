#include "hip/hip_runtime.h"
#include "CUDA.hpp"

/* change spherical vector into normalized xyz vector*/
__forceinline__ __host__ __device__ vec3f sphere_to_normal(const vec3f &sphere_direction)
{
    const float &theta = sphere_direction.y;
    const float &phi = sphere_direction.z;
    return vec3f(cosf(theta) * sinf(phi),
                 sinf(theta) * sinf(phi),
                 cosf(phi));
}

/* change xyz vector into spherical vector with 0 length */
// __forceinline__ __host__ __device__ vec3f normal_to_sphere(const vec3f &xyz_direction)
// {
//     float theta = atan(xyz_direction.y / xyz_direction.x);
//     float phi = acos(xyz_direction.z / length(xyz_direction));
//     if (xyz_direction.x < 0)
//     {
//         theta = (xyz_direction.y > 0) ? theta + M_PI : theta - M_PI;
//     }
//     return vec3f(0.f, theta, phi);
// }

/* change screen space position into ray direction */
__forceinline__ __host__ __device__ vec3f screen_to_direction(const vec2f &screen,
                                                              const vec3f &direction,
                                                              const vec3f &horizontal,
                                                              const vec3f &vertical)
{
    return normalize(direction + (screen.x - 0.5f) * horizontal + (screen.y - 0.5f) * vertical);
}

/* change polor into xy coordinate vector */
__forceinline__ __host__ __device__ vec2f polor_to_normal(const float &theta, const float &r)
{
    return vec2f(cosf(theta) * r, sinf(theta) * r);
}

/* make ray_stencil[] into an array stored directions of a ray stencil.
*ray_stencil: pointer of a vec2f ray directions array (screen space),
center_pixel_location: location of center pixel (screen space),
h: radius (screen space),
N: number of circles,
n: number of rays in first circle, must be multiple of 4. number doubles each time on next circle */
__host__ __device__ void makeRayStencil1(vec2f *const ray_stencil,
                                         const vec2f &center_pixel_location,
                                         const float &h,
                                         const int &N,
                                         const int &n)
{
    assert(h > 0.f);
    assert(N > 0);
    assert(n > 0 && n % 4 == 0);

    float temp_r;
    int temp_n{n};
    float theta;
    int index{0};
    for (int i = 0; i < N; i++)
    {
        temp_r = h / N * (i + 1);
        for (int j = 0; j < temp_n; j++)
        {
            theta = 2 * M_PI * j / temp_n;
            ray_stencil[index + j] = polor_to_normal(temp_r, theta) + center_pixel_location;
        }
        index += temp_n;
        temp_n *= 2;
    }
}

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__fastRenderer()
{
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;

    const int &numPixelSamples = optixLaunchParams.parameters.NUM_PIXEL_SAMPLES;
    const int &accumID = optixLaunchParams.frame.accumID;
    const auto &camera = optixLaunchParams.camera;

    PRD prd;
    prd.random.init(ix + accumID * optixLaunchParams.frame.size.x,
                    iy + accumID * optixLaunchParams.frame.size.y);
    prd.pixelColor = vec3f(255.f);

    // the values we store the PRD pointer in:
    uint32_t u0, u1;
    packPointer(&prd, u0, u1);

    vec3f pixelColor = 0.f;
    for (int sampleID = 0; sampleID < numPixelSamples; sampleID++)
    {
        vec3f rayDir;
        if (camera.camera_type == PINHOLE)
        {
            // normalized screen plane position, in [0,1]^2
            vec2f screen;
            if (numPixelSamples > 1)
            {
                screen = (vec2f(ix + prd.random(), iy + prd.random()) / vec2f(optixLaunchParams.frame.size));
            }
            else
            {
                screen = (vec2f(ix + 0.5f, iy + 0.5f) / vec2f(optixLaunchParams.frame.size));
            }

            // generate ray direction
            rayDir = screen_to_direction(screen, camera.direction, camera.horizontal, camera.vertical);
        }
        else if (camera.camera_type == ENV)
        {
            // sperical coordinate position
            vec3f spherical_position;
            if (numPixelSamples > 1)
            {
                spherical_position = ((ix + prd.random()) * camera.horizontal + (iy + prd.random()) * camera.vertical);
            }
            else
            {
                spherical_position = ((ix + 0.5f) * camera.horizontal + (iy + 0.5f) * camera.vertical);
            }
            spherical_position -= vec3f(0.f, M_PI, 0.f);
            // change into xyz coordinate position
            const vec3f xyz_position(sphere_to_normal(spherical_position));
            // view port transform
            rayDir = {dot(camera.matrix.vx, xyz_position),
                      dot(camera.matrix.vy, xyz_position),
                      dot(camera.matrix.vz, xyz_position)};
        }

        const int &ray_type = optixLaunchParams.parameters.LAUNCH_RAY_TYPE;
        optixTrace(optixLaunchParams.traversable,
                   camera.position,
                   rayDir,
                   0.f,   // tmin
                   1e20f, // tmax
                   0.0f,  // rayTime
                   OptixVisibilityMask(255),
                   OPTIX_RAY_FLAG_DISABLE_ANYHIT, //OPTIX_RAY_FLAG_NONE,
                   ray_type,                      // SBT offset
                   RAY_TYPE_COUNT,                // SBT stride
                   ray_type,                      // missSBTIndex
                   u0, u1);
        pixelColor += prd.pixelColor;
    }

    const int r = int(255.99f * min(pixelColor.x / numPixelSamples, 1.f));
    const int g = int(255.99f * min(pixelColor.y / numPixelSamples, 1.f));
    const int b = int(255.99f * min(pixelColor.z / numPixelSamples, 1.f));

    // convert to 32-bit rgba value (we explicitly set alpha to 0xff
    // to make stb_image_write happy ...
    const uint32_t rgba = 0xff000000 | (r << 0) | (g << 8) | (b << 16);

    // and write to frame buffer ...
    const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x;
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}

extern "C" __global__ void __raygen__classicRenderer()
{
    const int &numPixelSamples = optixLaunchParams.parameters.NUM_PIXEL_SAMPLES;
    const auto &camera = optixLaunchParams.camera;
    const int &N = optixLaunchParams.parameters.RAY_STENCIL_QUALITY.x;
    const int &n = optixLaunchParams.parameters.RAY_STENCIL_QUALITY.y;
    const float &radius = optixLaunchParams.parameters.RAY_STENCIL_RADIUS;

    // parameter check
    assert(radius > 0.f);
    assert(N > 0);
    assert(n > 0 && n % 4 == 0);

    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;

    // normalized screen plane position, in [0,1]^2
    vec2f screen = (vec2f(ix + 0.5f, iy + 0.5f) / vec2f(optixLaunchParams.frame.size));

    float temp_r;
    int temp_n{n};
    float theta;
    int index{0};
    for (int i = 0; i < N; i++)
    {
        temp_r = radius / N * (i + 1);
        for (int j = 0; j < temp_n; j++)
        {
            theta = 2 * M_PI * j / temp_n;
            vec3f rayDir = screen_to_direction(polor_to_normal(temp_r, theta) + screen, camera.direction, camera.horizontal, camera.vertical);

            PRD_Classic prd_classic;
            // the values we store the PRD pointer in:
            uint32_t u0, u1;
            packPointer(&prd_classic, u0, u1);
            optixTrace(optixLaunchParams.traversable,
                       camera.position,
                       rayDir,
                       0.f,   // tmin
                       1e20f, // tmax
                       0.0f,  // rayTime
                       OptixVisibilityMask(255),
                       OPTIX_RAY_FLAG_DISABLE_ANYHIT, //OPTIX_RAY_FLAG_NONE,
                       CLASSIC_RAY_TYPE,              // SBT offset
                       RAY_TYPE_COUNT,                // SBT stride
                       CLASSIC_RAY_TYPE,              // missSBTIndex
                       u0, u1);
            // if (ix == 500 & iy == 500)
            // {
            //     printf("%d\n", prd_classic.primID);
            // }
        }
        index += temp_n;
        temp_n *= 2;
    }

    // vec3f rayDir = screen_to_direction(screen, camera.direction, camera.horizontal, camera.vertical);

    // just try to calculate (2^N - 1) * n, cuda seems doesn't suppor pow(int, int)
    // int array_size{1};
    // for (int i = 0; i < quality.x; i++)
    // {
    //     array_size *= 2;
    // }
    // array_size -= 1;
    // array_size *= quality.y;

    // vec2f *rayStencil;
    // rayStencil = new vec2f[array_size];
    // makeRayStencil1(rayStencil, screen, radius, quality.x, quality.y);

    // // generate ray direction
    // vec3f *rayDir;
    // rayDir = new vec3f[array_size + 1];
    // for (int i = 0; i < array_size; i++)
    // {
    //     rayDir[i] = screen_to_direction(rayStencil[i], camera.direction, camera.horizontal, camera.vertical);
    // }
    // rayDir[array_size] = screen_to_direction(screen, camera.direction, camera.horizontal, camera.vertical);

    // // to do: ray tracing..

    // delete[] rayStencil;
    // delete[] rayDir;

    vec3f pixelColor = {0.f, 255.f, 0.f};
    const int r = int(255.99f * min(pixelColor.x / numPixelSamples, 1.f));
    const int g = int(255.99f * min(pixelColor.y / numPixelSamples, 1.f));
    const int b = int(255.99f * min(pixelColor.z / numPixelSamples, 1.f));

    // convert to 32-bit rgba value (we explicitly set alpha to 0xff
    // to make stb_image_write happy ...
    const uint32_t rgba = 0xff000000 | (r << 0) | (g << 8) | (b << 16);

    // and write to frame buffer ...
    const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x;
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}