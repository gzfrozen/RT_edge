#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "CUDA.hpp"

typedef struct
{
    float r; // a fraction between 0 and 1
    float g; // a fraction between 0 and 1
    float b; // a fraction between 0 and 1
} RGB;

typedef struct
{
    float h; // angle in degrees
    float s; // a fraction between 0 and 1
    float v; // a fraction between 0 and 1
} HSV;

__forceinline__ __host__ __device__ RGB hsv2rgb(const HSV &in)
{
    float hh, p, q, t, ff;
    int i;
    RGB out;

    if (in.s <= 0.0)
    { // < is bogus, just shuts up warnings
        out.r = in.v;
        out.g = in.v;
        out.b = in.v;
        return out;
    }
    hh = in.h;
    if (hh >= 360.0)
        hh = 0.0;
    hh /= 60.0;
    i = (int)hh;
    ff = hh - i;
    p = in.v * (1.0 - in.s);
    q = in.v * (1.0 - (in.s * ff));
    t = in.v * (1.0 - (in.s * (1.0 - ff)));

    switch (i)
    {
    case 0:
        out.r = in.v;
        out.g = t;
        out.b = p;
        break;
    case 1:
        out.r = q;
        out.g = in.v;
        out.b = p;
        break;
    case 2:
        out.r = p;
        out.g = in.v;
        out.b = t;
        break;

    case 3:
        out.r = p;
        out.g = q;
        out.b = in.v;
        break;
    case 4:
        out.r = t;
        out.g = p;
        out.b = in.v;
        break;
    case 5:
    default:
        out.r = in.v;
        out.g = p;
        out.b = q;
        break;
    }
    return out;
}

// calculate the distance from point P to line AB
__forceinline__ __host__ __device__ float point_to_line(const vec3f &AP, const vec3f &AB)
{
    return length(cross(AP, AB)) / length(AB);
}
//------------------------------------------------------------------------------
// closest hit programs.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but in some simple situations,
// only dummy programms is needed (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------

extern "C" __global__ void __closesthit__radiance()
{
    const TriangleMeshSBTData &sbtData = *(const TriangleMeshSBTData *)optixGetSbtDataPointer();
    PRD &prd = *getPRD<PRD>();

    // ------------------------------------------------------------------
    // gather some basic hit information
    // ------------------------------------------------------------------
    const int primID = optixGetPrimitiveIndex();
    const vec3i index = sbtData.index[primID];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // ------------------------------------------------------------------
    // compute normal, using either shading normal (if avail), or
    // geometry normal (fallback)
    // ------------------------------------------------------------------
    const vec3f &A = sbtData.vertex[index.x];
    const vec3f &B = sbtData.vertex[index.y];
    const vec3f &C = sbtData.vertex[index.z];
    vec3f Ng = cross(B - A, C - A);
    vec3f Ns = (sbtData.normal)
                   ? ((1.f - u - v) * sbtData.normal[index.x] + u * sbtData.normal[index.y] + v * sbtData.normal[index.z])
                   : Ng;

    // ------------------------------------------------------------------
    // face-forward and normalize normals
    // ------------------------------------------------------------------
    const vec3f rayDir = optixGetWorldRayDirection();

    if (dot(rayDir, Ng) > 0.f)
        Ng = -Ng;
    Ng = normalize(Ng);

    if (dot(Ng, Ns) < 0.f)
        Ns -= 2.f * dot(Ng, Ns) * Ng;
    Ns = normalize(Ns);

    // ------------------------------------------------------------------
    // compute diffuse material color, including diffuse texture, if
    // available
    // ------------------------------------------------------------------
    vec3f diffuseColor = sbtData.color;
    if (sbtData.hasTexture && sbtData.texcoord)
    {
        const vec2f tc = (1.f - u - v) * sbtData.texcoord[index.x] + u * sbtData.texcoord[index.y] + v * sbtData.texcoord[index.z];
        vec4f fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
        diffuseColor *= (vec3f)fromTexture;
    }

    // start with some ambient term
    vec3f pixelColor = (0.1f + 0.2f * fabsf(dot(Ns, rayDir))) * diffuseColor;

    // ------------------------------------------------------------------
    // compute shadow
    // ------------------------------------------------------------------
    const vec3f surfPos = (1.f - u - v) * sbtData.vertex[index.x] + u * sbtData.vertex[index.y] + v * sbtData.vertex[index.z];
    const int numLightSamples = optixLaunchParams.parameters.NUM_LIGHT_SAMPLES;
    for (int lightSampleID = 0; lightSampleID < numLightSamples; lightSampleID++)
    {
        // produce random light sample
        const vec3f lightPos = optixLaunchParams.light.origin + prd.random() * optixLaunchParams.light.du + prd.random() * optixLaunchParams.light.dv;
        vec3f lightDir = lightPos - surfPos;
        float lightDist = gdt::length(lightDir);
        lightDir = normalize(lightDir);

        // trace shadow ray:
        const float NdotL = dot(lightDir, Ns);
        if (NdotL >= 0.f)
        {
            vec3f lightVisibility = 0.f;
            // the values we store the PRD pointer in:
            uint32_t u0, u1;
            packPointer(&lightVisibility, u0, u1);
            optixTrace(optixLaunchParams.traversable,
                       surfPos + 1e-3f * Ng,
                       lightDir,
                       1e-3f,                     // tmin
                       lightDist * (1.f - 1e-3f), // tmax
                       0.0f,                      // rayTime
                       OptixVisibilityMask(255),
                       // For shadow rays: skip any/closest hit shaders and terminate on first
                       // intersection with anything. The miss shader is used to mark if the
                       // light was visible.
                       OPTIX_RAY_FLAG_DISABLE_ANYHIT | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
                       SHADOW_RAY_TYPE, // SBT offset
                       RAY_TYPE_COUNT,  // SBT stride
                       SHADOW_RAY_TYPE, // missSBTIndex
                       u0, u1);
            pixelColor += lightVisibility * optixLaunchParams.light.power * diffuseColor * (NdotL / (lightDist * lightDist * numLightSamples));
        }
    }

    prd.pixelColor = pixelColor;
}

extern "C" __global__ void __closesthit__shadow()
{
    /* not going to be used ... */
}

extern "C" __global__ void __closesthit__phase()
{
    const TriangleMeshSBTData &sbtData = *(const TriangleMeshSBTData *)optixGetSbtDataPointer();
    PRD &prd = *getPRD<PRD>();

    // ------------------------------------------------------------------
    // phase and color calculation
    // ------------------------------------------------------------------
    const float distance = optixGetRayTmax();
    const float phase = fmod(distance, optixLaunchParams.parameters.WAVE_LENGTH) * 360.f / optixLaunchParams.parameters.WAVE_LENGTH;
    const HSV hsv = {phase, 1.f, 0.7f}; // use hsv color space
    const RGB rgb = hsv2rgb(hsv);

    // ------------------------------------------------------------------
    // gather some basic hit information
    // ------------------------------------------------------------------
    const int primID = optixGetPrimitiveIndex();
    const vec3i index = sbtData.index[primID];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // ------------------------------------------------------------------
    // compute normal, using either shading normal (if avail), or
    // geometry normal (fallback)
    // ------------------------------------------------------------------
    const vec3f &A = sbtData.vertex[index.x];
    const vec3f &B = sbtData.vertex[index.y];
    const vec3f &C = sbtData.vertex[index.z];
    vec3f Ng = cross(B - A, C - A);
    vec3f Ns = (sbtData.normal)
                   ? ((1.f - u - v) * sbtData.normal[index.x] + u * sbtData.normal[index.y] + v * sbtData.normal[index.z])
                   : Ng;

    // ------------------------------------------------------------------
    // face-forward and normalize normals
    // ------------------------------------------------------------------
    const vec3f rayDir = optixGetWorldRayDirection();

    if (dot(rayDir, Ng) > 0.f)
        Ng = -Ng;
    Ng = normalize(Ng);

    if (dot(Ng, Ns) < 0.f)
        Ns -= 2.f * dot(Ng, Ns) * Ng;
    Ns = normalize(Ns);

    // ------------------------------------------------------------------
    // final result mixed with some simple ambient effect
    // ------------------------------------------------------------------
    prd.pixelColor = (0.1f + 0.2f * fabsf(dot(Ns, rayDir))) + vec3f(rgb.r, rgb.g, rgb.b);
}

extern "C" __global__ void __closesthit__mono()
{
    const TriangleMeshSBTData &sbtData = *(const TriangleMeshSBTData *)optixGetSbtDataPointer();
    PRD &prd = *getPRD<PRD>();

    // ------------------------------------------------------------------
    // gather some basic hit information
    // ------------------------------------------------------------------
    const int primID = optixGetPrimitiveIndex();
    const vec3i index = sbtData.index[primID];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // ------------------------------------------------------------------
    // compute normal, using either shading normal (if avail), or
    // geometry normal (fallback)
    // ------------------------------------------------------------------
    const vec3f &A = sbtData.vertex[index.x];
    const vec3f &B = sbtData.vertex[index.y];
    const vec3f &C = sbtData.vertex[index.z];

    const vec3f AB = B - A;
    const vec3f BC = C - B;
    const vec3f CA = A - C;

    vec3f Ng = cross(CA, AB);
    vec3f Ns = (sbtData.normal)
                   ? ((1.f - u - v) * sbtData.normal[index.x] + u * sbtData.normal[index.y] + v * sbtData.normal[index.z])
                   : Ng;

    // ------------------------------------------------------------------
    // face-forward and normalize normals
    // ------------------------------------------------------------------
    const vec3f rayDir = optixGetWorldRayDirection();

    if (dot(rayDir, Ng) > 0.f)
        Ng = -Ng;
    Ng = normalize(Ng);

    if (dot(Ng, Ns) < 0.f)
        Ns -= 2.f * dot(Ng, Ns) * Ng;
    Ns = normalize(Ns);

    // ------------------------------------------------------------------
    // compute edge
    // ------------------------------------------------------------------
    // edge ray start position
    const vec3f surfPos = (1.f - u - v) * sbtData.vertex[index.x] + u * sbtData.vertex[index.y] + v * sbtData.vertex[index.z];
    const vec3f surfDepth = optixLaunchParams.parameters.EDGE_DETECTION_DEPTH * Ng;

    // edge ray direction
    const vec3f edge_direction[3] = {cross(AB, Ng), cross(BC, Ng), cross(CA, Ng)};

    // distance from hit position to triangle edge
    const float edge_distance[3] = {
        point_to_line(surfPos - A, AB),
        point_to_line(surfPos - B, BC),
        point_to_line(surfPos - C, CA)};

    bool is_edge = false;
    // trace edge ray
    for (int i = 0; i < 3; i++)
    {
        // per ray date for edge detection
        PRD_Edge prd_edge;
        if (edge_distance[i] > optixLaunchParams.parameters.MAX_EDGE_DISTANCE)
            continue;
        prd_edge.edge_distance = edge_distance[i];
        prd_edge.is_edge = false;
        // the values we store the PRD_Edge pointer in:
        uint32_t u0, u1;
        packPointer(&prd_edge, u0, u1);
        optixTrace(optixLaunchParams.traversable,
                   surfPos - surfDepth,
                   edge_direction[i],
                   0,                         // tmin
                   edge_distance[i] * 1.001f, // tmax
                   0.0f,                      // rayTime
                   OptixVisibilityMask(255),
                   OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                   EDGE_RAY_TYPE,  // SBT offset
                   RAY_TYPE_COUNT, // SBT stride
                   EDGE_RAY_TYPE,  // missSBTIndex
                   u0, u1);
        prd_edge.is_edge ? is_edge = true : 0;
    }
    is_edge ? prd.pixelColor = {0.f, 0.f, 0.f} : 0;
}

extern "C" __global__ void __closesthit__edge()
{
    const float hit_distance = optixGetRayTmax();
    PRD_Edge &prd_edge = *getPRD<PRD_Edge>();
    float x = prd_edge.edge_distance - hit_distance;
    float edge_angle;
    edge_angle = atan2f(optixLaunchParams.parameters.EDGE_DETECTION_DEPTH, x);
    // printf("%f\n", edge_angle);
    edge_angle <= optixLaunchParams.parameters.MAX_EDGE_ANGLE ? prd_edge.is_edge = true : 0;
}